#include "hip/hip_runtime.h"
#include "AprioriRule.h"
#include <thrust/device_vector.h>

AprioriRule::~AprioriRule() {
	hipFree(_transactions_);
	hipFree(_lCount_);
}
AprioriRule::AprioriRule(vector<int> _transactions, vector<int> _product, int tPeerB, int Cores, long double _minSp, long double _minConfi)
{
	Step = 0;
	minSuport = _minSp;
	minConfin = _minConfi;
	Product = _product;
	defineThread = tPeerB;
	defineCores = Cores -1;
	peerSize = _product.size();
	frequentItemsets.push_back({ {} });
	tranSize = _transactions.size()/peerSize;

	// int *tempTrans = new int[peerSize*tranSize];
	_countC = new int[tranSize];
	// for (int i = 0; i < tranSize; i++) {
	// 	// Parallel here
	// 	_countC[i] = 0;
	// 	for (int j = 0; j < peerSize; j++) {
	// 		tempTrans[i*peerSize + j] = _transactions[i][j];
	// 	}
	// }
	hipMalloc(&_transactions_, sizeof(int)*(peerSize*tranSize));
	hipMalloc(&_lCount_, sizeof(int)*tranSize);
	hipMemcpy(_transactions_, _transactions.data(), sizeof(int)*(peerSize*tranSize), hipMemcpyHostToDevice);
	hipMemcpy(_lCount_, _countC, sizeof(int)*(tranSize), hipMemcpyHostToDevice);

	_transactions.clear();
	printf("%f\tLoad Data\n", clock() - startTime);

	// delete tempTrans; tempTrans = NULL;
}
vector<tuple<vector<int>, vector<int>, long double, long double>> AprioriRule::getResult()
{
	return AprioriRules;
}
void AprioriRule::Process()
{
	for (auto i = Product.begin(); i != Product.end(); i++)
		C.push_back(vector<int>(1, *i));
	Step++;
	L = generateNewL();
	frequentItemsets.push_back(L);

	while (true)
	{
		printf("--Step %d -> %f\n", Step, clock() - startTime);
		C = generateNextC();
		if (C.size() == 0)
			break;
		printf("%d\t%d\n",C.size(),L.size());
		Step++;
		L = generateNewL();

		frequentItemsets.push_back(L);
	}
	printf("Start check frequent Itemset %f\n", clock() - startTime);
	int fSize = frequentItemsets.size();
	int *peers = new int[defineCores+1];
	int peer = fSize / (defineCores);
	for (int i =0 ; i < defineCores; i++) peers[i] = peer;
	int mod = fSize % (defineCores);
	peers[defineCores] = ((mod == 0)? 0 : mod);
	thread *mThread = new thread[defineCores+1];
	for (int i = 0; i <= defineCores; i++) {
		mThread[i] = thread(&AprioriRule::parallelFrequent, this, i*peer, peers[i]);
	}
	for (int i = 0; i <= defineCores; i++) {
		mThread[i].join();
	}
	printf("Start Write output %f\n", clock() - startTime);
}

void AprioriRule::parallelFrequent(int start, int loop)
{
	loop = start + loop;
	for (int i = start; i < loop; i++) {
		// Parallel here
		for (auto &items : frequentItemsets[i])
		{
			generateAssociationRule(items, {}, {}, 0, minConfin);
		}
	}
}

void AprioriRule::generateAssociationRule(vector<int> items, vector<int> X, vector<int> Y, int index, long double minConfi) {
	if (index == items.size()) {
		if (X.size() == 0 || Y.size() == 0) return;
		long double XYsupport = getSupport(getIndex(items));
		long double Xsupport = getSupport(getIndex(X));

		if (Xsupport == 0) return;

		long double support = (long double)XYsupport;
		long double confidence = (long double)XYsupport / Xsupport * 100.0;
		if (confidence >= minConfi){
			Block.lock();
			AprioriRules.push_back({ X, Y, support, confidence });
			Block.unlock();
		}
		return;
	}

	X.push_back(items[index]);
	generateAssociationRule(items, X, Y, index + 1, minConfi);
	X.pop_back();
	Y.push_back(items[index]);
	generateAssociationRule(items, X, Y, index + 1, minConfi);
}

void AprioriRule::generateCParallel1(int start, int loop, int Lsize, vector<vector<int>>& temp)
{
	loop = start + loop;
	for (int i = start; i < loop; i++) {
		for (int j = i + 1; j < Lsize; j++) {
			int k;
			for (k = 0; k < Step - 1; k++)
				if (L[i][k] != L[j][k]) break;
			if (k == Step - 1) {
				vector<int> ret;
				for (int h = 0; h < Step - 1; h++)
					ret.push_back(L[i][h]);
				int f = L[i][Step - 1];
				int s = L[j][Step - 1];
				if (f > s) swap(f, s);
				Block.lock();
				ret.push_back(f), ret.push_back(s);
				temp.push_back(ret);
				Block.unlock();
			}
		}
	}
}

void AprioriRule::generateCParallel2(int start, int loop, set<vector<int>> LSet, vector<vector<int>>& temps, vector<vector<int>> temp)
{
	loop = start + loop;
	for (int l = start; l < loop; l++) {
		//MAYBE
		int i, size = temp[l].size();
		for (i = 0; i < size; i++) {
			vector<int> tmp = temp[l];
			tmp.erase(tmp.begin() + i);
			if (LSet.find(tmp) == LSet.end())
				break;
		}
		if (i == size) {
			Block.lock();
			temps.push_back(temp[l]);
			Block.unlock();
		}
	}
}

vector<vector<int>> AprioriRule::generateNextC() {
	printf("---Generate C:\t%f \t->", clock() - startTime);
	vector<vector<int>> temp;
	int Lsize = L.size();
	int *peers = new int[defineCores+1];
	int peer = Lsize/defineCores;
	for (int i = 0; i < defineCores; i++){
		peers[i] = peer;
	}
	int mod = Lsize%defineCores;
	peers[defineCores] =   ((mod != 0) ? mod : 0);

	thread *mThread = new thread[defineCores+1];
	for (int i = 0; i <= defineCores; i++) {
		mThread[i] = thread(&AprioriRule::generateCParallel1, this, i*peer, peers[i], Lsize, ref(temp));
	}
	for (int i = 0; i <= defineCores; i++) {
		mThread[i].join();
	}
	vector<vector<int>> temps;
	set<vector<int>> LSet;
	for (auto&row : L)
		LSet.insert(row);

	int Tsize = temp.size();
	peer = Tsize / (defineCores);
	for (int i = 0; i < defineCores; i++){
		peers[i] = peer;
	}
	mod = Tsize % (defineCores);
	peers[defineCores] =   ((mod != 0) ? mod : 0);
	for (int i = 0; i <= defineCores; i++) {
		mThread[i] = thread(&AprioriRule::generateCParallel2, this,i*peer,peers[i], LSet, ref(temps), temp);
	}
	for (int i = 0; i <= defineCores; i++) {
		mThread[i].join();
	}
	printf("%f\n", clock() - startTime);
	delete peers; peers = NULL;
	return temps;
}
vector<vector<int>> AprioriRule::generateNewL() {
	printf("---Generate L: %f \t ->", clock() - startTime);
	vector<vector<int>> ret;
	int Csize = C.size();
	int *peers = new int[defineCores+1];
	int peer = Csize/defineCores;
	for (int i = 0; i < defineCores; i++){
		peers[i] = peer;
	}	
	int mod = Csize%defineCores;
	peers[defineCores] = ((mod != 0) ? mod: 0);
	thread *mThread = new thread[defineCores+1];
	for (int i = 0; i <= defineCores; i++){
		mThread[i] = thread(&AprioriRule::generateLParallel,this,i*peer,peers[i], ref(ret));
	}
	for (int i = 0; i <= defineCores; i++){
		mThread[i].join();
	}
	delete peers; peers = NULL;
	printf("%f\n", clock() - startTime);
	return ret;
}

void AprioriRule::generateLParallel(int start,int loop, vector<vector<int>> &ret){
	loop = start + loop;
	for (int  i = start; i < loop; i ++){
		long double sp = getSupport(getIndex(C[i]));
		if (sp < minSuport) continue;
		Block.lock();
		ret.push_back(C[i]);
		Block.unlock();
	}
}

vector<int> AprioriRule::getIndex(vector<int> row) {
	vector<int> temp;

	for (int i = 0, j = 0; j < row.size(); i++) {
		if (Product[i] == row[j]) {
			temp.push_back(i);
			j++;
		}
	}
	return temp;
}


__global__ void ItemSupport(int *count_, int* _transactions_, int* item, int itemSize, int peerSize) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	bool boo;
	boo = true;
	for (int i = 0; i < itemSize; i++) {
		if (_transactions_[peerSize*(index)+item[i]] == 0) {
			boo = false;
			break;
		}
	}
	count_[index] = (boo ? 1: 0);
}

long double AprioriRule::getSupport(vector<int> item) {
	int count = 0;
	int *item_, itemSize = item.size();
	Block.lock();
	hipMalloc(&item_, sizeof(int)*itemSize);
	hipMemcpy(item_, item.data(), sizeof(int)*itemSize, hipMemcpyHostToDevice);
	Block.unlock();
	if (tranSize < defineThread) {
		ItemSupport << <1, tranSize >> > (_lCount_, _transactions_, item_, itemSize, peerSize);
	}
	else {
		// <<< Blocks , Threads >>>
		ItemSupport << <tranSize / defineThread, defineThread >> > (_lCount_, _transactions_, item_, itemSize, peerSize);
	}
	hipFree(item_);

	hipMemcpy(_countC, _lCount_, sizeof(int)*tranSize, hipMemcpyDeviceToHost);
	for (int i = 0; i < tranSize; i++) {
		if (_countC[i] == 1) count++;
	}
	return (long double)count / tranSize * 100;
}